#include "hip/hip_runtime.h"
%%writefile idx_cal2d.cu

#include<stdio.h>


__global__ void unique_gid_calculation_2d(int* data)
{
  int tid = threadIdx.x;
  int block_offset = blockIdx.x*blockDim.x;
  int row_offset = blockDim.x * gridDim.x * blockIdx.y;
  int gid = row_offset + block_offset + tid;
  printf("blockIdx: %d, blockIdx.y: %d tid: %d, gid: %d, data: %d\n",
  blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

int main()
{
  int arr_size = 16;
  int array_byte_size = sizeof(int) * arr_size;
  int h_data[] = {23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};
  for(int i = 0; i < arr_size; i++)
  {
    printf("h_data[%d]: %d\n", i, h_data[i]);
  }
  printf("\n \n");

  int* d_data;
  hipMalloc((void**)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 block(4);
  dim3 grid(2,2);

  unique_gid_calculation_2d<<<grid, block>>>(d_data);
  hipDeviceSynchronize();

  hipDeviceReset();


  return 0;
}