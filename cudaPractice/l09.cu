#include "hip/hip_runtime.h"
%%writefile time_measure.cu

#include<stdio.h>


__global__ void sum_array_gpu(int* a, int* b, int* c, int size)
{
  int gid = blockIdx.x*blockDim.x + threadIdx.x;
  if (gid < size) {
    c[gid] = a[gid] + b[gid];
  }
}

void sum_array_cpu(int* a, int* b, int* c, int size)
{
  for(int i = 0; i < size; i++)
  {
    c[i] = a[i] + b[i];
  }
}

void cmp_array(int* a, int* b, int size)
{
  for(int i =0; i < size; i++)
  {
    if (a[i] != b[i]) {
      printf("Array are different\n");
      return;
    }
  }
  printf("Array are same\n");
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

int main()
{
  int size = 1<<25;
  int block_size = 256;

  int NO_BYTES = size*sizeof(int);

  int* h_a, *h_b, *gpu_results, *h_c;

  h_a = (int*)malloc(NO_BYTES);
  h_b = (int*)malloc(NO_BYTES);
  h_c = (int*)malloc(NO_BYTES);
  gpu_results = (int*)malloc(NO_BYTES);

  time_t t;
  srand((unsigned)time(&t));
  for(int i = 0; i < size; i++)
  {
    h_a[i] = (int)(rand()&0xff);
  }

  for(int i = 0; i < size; i++)
  {
    h_b[i] = (int)(rand()&0xff);
  }
  clock_t cpu_start, cpu_end;
  cpu_start = clock();
  sum_array_cpu(h_a, h_b, h_c, size);
  cpu_end = clock();


  memset(gpu_results,0,NO_BYTES);

  int* d_a, *d_b, *d_c;
  hipMalloc((int**)&d_a, NO_BYTES);
  
  hipMalloc((int**)&d_b, NO_BYTES);
 
  hipMalloc((int**)&d_c, NO_BYTES);
 
  clock_t htod_start, htod_end;
  htod_start = clock();
  hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
  htod_end = clock();
  
  dim3 block(block_size);
  dim3 grid((size/block.x)+1);

  clock_t gpu_start, gpu_end;
  gpu_start = clock();
  sum_array_gpu<<<grid,block>>>(d_a, d_b, d_c, size);
  hipDeviceSynchronize();
  gpu_end = clock();


  clock_t dtoh_start, dtoh_end;
  dtoh_start = clock();
  hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
  dtoh_end = clock();
  cmp_array(gpu_results, h_c, size);

  printf("Sum array CPU execution time: %4.6f \n",
    (cpu_end-cpu_start)/(double)CLOCKS_PER_SEC);
  printf("Sum array GPU execution time: %4.6f \n",
    (gpu_end-gpu_start)/(double)CLOCKS_PER_SEC);
  printf("Host to device transfer time: %4.6f \n",
    (htod_end-htod_start)/(double)CLOCKS_PER_SEC);
  printf("Device to host transfer time: %4.6f \n",
    (dtoh_end-dtoh_start)/(double)CLOCKS_PER_SEC);
  printf("Sum array GPU total execution time: %4.6f \n",
    (dtoh_end-htod_start)/(double)CLOCKS_PER_SEC);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(gpu_results);

  hipDeviceReset();

  return 0;
}