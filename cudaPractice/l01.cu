#include "hip/hip_runtime.h"
%%writefile hello_cuda.cu

#include<stdio.h>

__global__ void hello_cuda()
{
  printf("hello CUDA world\n");
}

int main()
{
  int nx,ny;
  nx = 16;
  ny = 4;
  dim3 block(8,2,1);
  dim3 grid(nx/block.x,ny/block.y,1);
  hello_cuda<<<grid,block>>>();
  hipDeviceSynchronize();

  hipDeviceReset();
  return 0;
}